
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

__global__ void myfunc(double *vals, size_t n) {
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprandState state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

    for (size_t i = 0; i < n; i++) {
        vals[i] = hiprand_uniform_double(&state);
    }
}

int main() {
    size_t n = 20;
    size_t num_bytes = n * sizeof(double);

    double *vals_host = 0;
    vals_host = (double*)malloc(num_bytes);
    double *vals_device = 0;
    hipMalloc((void**)&vals_device, num_bytes);

    myfunc<<<1, 1>>>(vals_device, n);

    hipMemcpy(vals_host, vals_device, num_bytes, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < n; i++) {
        printf("%d = %lf\n", i, vals_host[i]);
    }

    free(vals_host);
    hipFree(vals_device);

    return 0;
}
