
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

// Adapted from
// https://stackoverflow.com/questions/26650391/generate-random-number-within-a-function-with-curand-without-preallocation
__global__ void myfunc(double *vals, size_t n) {
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprandState state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

    for (size_t i = 0; i < n; i++) {
        vals[i] = hiprand_uniform_double(&state);
    }
}

// Adapted from
// https://stackoverflow.com/questions/7989039/use-of-cudamalloc-why-the-double-pointer
int main() {
    size_t n = 20;
    size_t num_bytes = n * sizeof(double);

    double *vals_host = 0;
    vals_host = (double*)malloc(num_bytes);
    double *vals_device = 0;
    hipMalloc((void**)&vals_device, num_bytes);

    myfunc<<<1, 1>>>(vals_device, n);

    hipMemcpy(vals_host, vals_device, num_bytes, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < n; i++) {
        printf("%d = %lf\n", i, vals_host[i]);
    }

    free(vals_host);
    hipFree(vals_device);

    return 0;
}
